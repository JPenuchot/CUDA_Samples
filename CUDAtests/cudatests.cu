#include <hip/hip_runtime.h>


#include <iostream>


// Kernel de test
__global__ void Testf(int Tab[], int *casesperthread)
{
	int Pos = blockIdx.x * blockDim.x + threadIdx.x;

	for(int i = Pos * *casesperthread; i < (Pos * *casesperthread) + *casesperthread; i++)
		Tab[i] = i;
}

int main()
{
	// Declarations
	int			blocks = 14,
				threads = 32,
				casesperthread = 400000,
				vsize = blocks * threads * casesperthread;
	
	int *d_casesperthread = &casesperthread;
	
	// CPU allocation block
	int *h_Tab = (int*)malloc(vsize * sizeof(int)),
		*d_Tab;

	// CUDA allocation block
	hipMalloc((void**)&d_Tab, vsize * sizeof(int));
	hipMalloc((void**)&d_casesperthread, sizeof(int));

	hipMemcpy(d_Tab, h_Tab, vsize * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_casesperthread, &casesperthread, sizeof(int), hipMemcpyHostToDevice);

	// Kernel invocation (448 threads)
	Testf <<<blocks, threads>>> (d_Tab, d_casesperthread);

	// CUDA free block
	//cudaMemcpy(h_Tab, d_Tab, vsize * sizeof(int), cudaMemcpyDeviceToHost);
	hipFree(d_Tab);
	hipFree(d_casesperthread);
	
	// CPU free block
	free(h_Tab);

	return 0;
}