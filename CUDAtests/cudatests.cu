#include "hip/hip_runtime.h"
#include <iostream>

//	This code sample isn't very useful, it writes the position of the cell for each cell in the table.
//
//	SUMARRY :
//
//	This sample was made to be as simple as possible.
//
//	Variable names :			- h_(varname)		= HOST (CPU) variable
//								- d_(varname)		= DEVICE (GPU) variable
//								- no prefix			= Local variables (They are not transferred between the CPU and the GPU inside the block)
//
//	CUDA variables/functions :	- blockIdx			= vec3 struct gives you the position of the block the kernel runs in
//								- threadIdx			= vec3 struct gives you the position of the thread inside the block the kernel runs in
//								- hipMalloc()		= Simillar to malloc(), but allocs memory on the graphics card
//								- cudaMemCpy()		= Used to copy content from GPU to CPU and vice versa
//

//	Kernel
__global__ void Testf(int Tab[], int *casesperthread)
{
	int Pos = blockIdx.x * blockDim.x + threadIdx.x;

	for(int i = Pos * *casesperthread; i < (Pos * *casesperthread) + *casesperthread; i++)
		Tab[i] = i;
}

//	Main
int main()
{
	//	Declarations
	int			blocks = 14,
				threads = 32,
				h_casesperthread = 400000,
				vsize = blocks * threads * h_casesperthread;

	//	Declarating casesperthread for the GPU
	int			*d_casesperthread = &h_casesperthread;
	
	//	Matrix allocation and declaration block
	int *h_Tab = (int*)malloc(vsize * sizeof(int)),
		*d_Tab;

	//	CUDA allocation block
	hipMalloc((void**)&d_Tab, vsize * sizeof(int));
	hipMalloc((void**)&d_casesperthread, sizeof(int));

	hipMemcpy(d_Tab, h_Tab, vsize * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_casesperthread, &casesperthread, sizeof(int), hipMemcpyHostToDevice);

	//	Kernel invocation (448 threads)
	Testf <<<blocks, threads>>> (d_Tab, d_casesperthread);

	//	CUDA free block
	hipMemcpy(h_Tab, d_Tab, vsize * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_Tab);
	hipFree(d_casesperthread);
	
	//	CPU free block
	free(h_Tab);

	return 0;
}