
#include <hip/hip_runtime.h>
#include <iostream>

//	Virtual grid dimensions :
//
//	20 000 * 20 000
//	20 blocks, 20 threads
//	2 000 cells/thread
//
//	Matrix dimensions :
//	
//	20*20 Matrix
//	Virtualized as a 400 cells array (20 * 20)
//	Each cell of the matrix has 2 000 virtual cells
//
//	Algorithm used to get a position into the thread grid :
//
//	Pos = blockIdx.x * blockDim.x + threadIdx.x


//	SUMARRY :
//	
//	Variables names :			- h_(varname)		= HOST (CPU) variable
//								- d_(varname)		= DEVICE (GPU) variable
//								- no prefix			= Local variables (They are not transferred between the CPU and the GPU inside the block)
//
//	Variables/Matrix usage :	- Mat[]				= The sum of each value of each cell of this virtual matrix gives the area of the quarter disk
//								- vCellsPerThread	= Number of virtual cells for each thread (One thread checks for several cells)
//								- MatX/MatY			= Matrix dimensions (Blocks * Threads)
//								- Area				= Sum of each value of each cell of Mat[] after computing
//								- Pi				= Approx value of Pi
//								- SQGridSize		= Stands for Square Grid Size, it is the area of the virtual grid (MatX * MatY * vCellsPerThread * vCellsPerThread)
//								- MatSize			= Size of the 400 cells matrix
//
//	CUDA variables/functions :	- blockIdx			= vec3 struct giving you the position of the block the kernel is located in
//								- threadIdx			= vec3 struct giving you the position of the thread inside the block the kernel is located in
//								- cudaMalloc()		= Simillar to malloc(), but allocs memory on the graphics card
//								- cudaMemCpy()		= Used to copy content from GPU to CPU and vice versa


// This is the kernel running on the GPU. It is pretty basic, it checks for each case if it is inside or outside the quarter disk
__global__ void MatComputing( double Mat[],  int *vCellsPerThread,  int *MatX,  int *MatY) {

	// Getting the position of the kernel and the grid size
	int	Pos = blockIdx.x * blockDim.x + threadIdx.x,
		SQGridSize = *MatX * *MatY * *vCellsPerThread * *vCellsPerThread;

	Mat[Pos] = 0;

	// This tests each cell to know if it is or not inside the quarter disk
	for(double i = blockIdx.x * *vCellsPerThread; i < (blockIdx.x * *vCellsPerThread) + *vCellsPerThread; i++)
		for(double j = threadIdx.x * *vCellsPerThread; j < (threadIdx.x * *vCellsPerThread) + *vCellsPerThread; j++)
			if((i * i) + (j * j) <= SQGridSize)
				Mat[Pos]++;
}

int main() {
	
	// DECLARATIONS AND ALLOCATIONS ON THE GPU
	const  int		h_MatX = 20,
					h_MatY = 20,
					h_vCellsPerThread = 2000;

	const size_t	MatSize = h_MatX * h_MatY * sizeof( double);
					
	double			*h_Mat = ( double*)malloc(MatSize),
					*d_Mat;

	int				*d_MatX = ( int*)malloc(sizeof( int)),
					*d_MatY = ( int*)malloc(sizeof( int)),
					*d_vCellsPerThread;
					

	double			Pi = 0,
					Area = 0;

	hipMalloc((void**)&d_Mat, MatSize);
	hipMalloc((void**)&d_vCellsPerThread, sizeof( double));
	hipMalloc((void**)&d_MatX, sizeof( int));
	hipMalloc((void**)&d_MatY, sizeof( int));

	hipMemcpy(d_Mat, h_Mat, MatSize, hipMemcpyHostToDevice);
	hipMemcpy(d_vCellsPerThread, &h_vCellsPerThread, sizeof( double), hipMemcpyHostToDevice);
	hipMemcpy(d_MatX, &h_MatX, sizeof( int), hipMemcpyHostToDevice);
	hipMemcpy(d_MatY, &h_MatY, sizeof( int), hipMemcpyHostToDevice);

	// MAIN PROGRAM
	MatComputing <<<h_MatX, h_MatY>>> (d_Mat, d_vCellsPerThread, d_MatX, d_MatY);

	hipMemcpy(h_Mat, d_Mat, MatSize, hipMemcpyDeviceToHost);

	for(int i = 0; i < h_MatX * h_MatY; i++)
		Area+= h_Mat[i];

	Pi = (Area * 4) / ((h_MatX * h_vCellsPerThread) * (h_MatY * h_vCellsPerThread));

	std::cout << Pi << std::endl;
	std::cin.get();

	// MEMORY DISALLOCATION
	hipFree(d_Mat);
	hipFree(d_vCellsPerThread);
	hipFree(d_MatX);
	hipFree(d_MatY);

	free(h_Mat);

	return 0;
}